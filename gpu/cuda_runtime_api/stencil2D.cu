//#define DOUBLE_PRECISION
#ifdef DOUBLE_PRECISION
// use this for double precision
//#pragma OPENCL EXTENSION cl_khr_fp64 : enable
typedef double real_t;
#else
typedef float real_t;
#endif

//crap: cannot separate C++ from CUDA
#include <hip/hip_runtime.h>

typedef unsigned uint;


//simple kernel for 2D stencils
/*extern "C"*/ __global__ void Stencil2D(
                         const real_t* dom,    // input
                         real_t* dom2,        // output
                         uint nrows, uint ncolumns,     // size
                         const real_t* stencil // filter      
                        )
{

//infinite loop produces the following behavior:
// -tesla:   nothing, computation goes on forever, program can
//           stopped with ^C
// -geforce: computation stops after a number of seconds; reading
//           data back results in an error
// -cpu:     compiler reports unreachable code and program
//           segfaults during kernel execution
//while(1);

    const int FILTER_SIZE = 3;
    const int IDX_BOUND = FILTER_SIZE >> 1;
    const int r = blockIdx.y * blockDim.y + threadIdx.y;
    const int c = blockIdx.x * blockDim.x + threadIdx.x;
    real_t sum = 0.0;
    for( int i = -IDX_BOUND; i <= IDX_BOUND; ++i )
    {
        int idxiS  = ( i + IDX_BOUND ) * FILTER_SIZE + IDX_BOUND;
        for ( int j = -IDX_BOUND; j <= IDX_BOUND; ++j )
        {
            int idxS = idxiS + j;
            int idxI = r + i;
            int idxJ = c + j;
            if( idxI < 0 ) idxI = nrows + i; //i < 0
            if( idxJ < 0 ) idxJ = ncolumns + j; //j < 0
            if( idxI >= nrows ) idxI -= nrows; //i > 0
            if( idxJ >= ncolumns ) idxJ -= ncolumns; // j > 0   
            sum += stencil[ idxS  ] * dom[ idxI * ncolumns + idxJ ];
        }
    }
    dom2[ r * ncolumns + c ] = sum;	
}

/*extern "C"*/ void RunStencil2D( dim3 b, dim3 tpb,
								  const real_t* dom, // input
								  real_t* dom2, // output
								  unsigned nrows, unsigned ncolumns,// size
								  const real_t* stencil // filter      
								)
{
	Stencil2D<<<b,tpb>>>(dom, dom2, nrows, ncolumns, stencil );
}